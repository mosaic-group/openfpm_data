//
// Created by tommaso on 4/07/19.
//
#define BOOST_TEST_DYN_LINK
#define DISABLE_MPI_WRITTERS

//#define SPARSEGRIDGPU_LAUNCH_BOUND_APPLY_STENCIL_IN_PLACE __launch_bounds__(512)
#define SPARSEGRIDGPU_LAUNCH_BOUND_APPLY_STENCIL_IN_PLACE_NO_SHARED __launch_bounds__(BLOCK_SIZE_STENCIL,12)

#include <boost/test/unit_test.hpp>
#include "SparseGridGpu/SparseGridGpu.hpp"
#include "cuda_macro.h"
#include "util/stat/common_statistics.hpp"
#include "Plot/GoogleChart.hpp"
#include "util/performance/performance_util.hpp"
#include "SparseGridGpu/tests/utils/SparseGridGpu_testKernels.cuh"
#include <set>
#include "performancePlots.hpp"
#include "SparseGridGpu/tests/utils/SparseGridGpu_util_test.cuh"

extern char * test_dir;

// Property tree

report_sparse_grid_tests report_sparsegrid_funcs;
std::string suiteURI = "performance.SparseGridGpu";
std::set<std::string> testSet;


BOOST_AUTO_TEST_SUITE(performance)

BOOST_AUTO_TEST_SUITE(SparseGridGpu_test)


template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeatGet_perf(unsigned int i, std::string base)
{
    auto testName = "In-place GET stencil";
    typedef HeatStencilGet<SparseGridZ::dims,0,1> Stencil01T;
    typedef HeatStencilGet<SparseGridZ::dims,1,0> Stencil10T;

    // typedef HeatStencilGet<SparseGridZ::dims,0,0> Stencil01T;
    // typedef HeatStencilGet<SparseGridZ::dims,0,0> Stencil10T;

    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*SparseGridZ::blockEdgeSize_);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize);
    dim3 blockSize(SparseGridZ::blockEdgeSize_,SparseGridZ::blockEdgeSize_);
    typename SparseGridZ::grid_info blockGeometry(gridSize);
    SparseGridZ sparseGrid(blockGeometry);
    gpu::ofp_context_t gpuContext;
    sparseGrid.template setBackgroundValue<0>(0);

    unsigned long long numElements = gridEdgeSize*SparseGridZ::blockEdgeSize_*gridEdgeSize*SparseGridZ::blockEdgeSize_;

    // Initialize the grid
    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
    sparseGrid.template flush < sRight_ < 0 >> (gpuContext, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    dim3 sourcePt(gridSize.x * SparseGridZ::blockEdgeSize_ / 2, gridSize.y * SparseGridZ::blockEdgeSize_ / 2, 0);
    insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
    sparseGrid.template flush < sRight_ < 0 >> (gpuContext, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

    iterations /= 2;
    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencils<Stencil01T>(sparseGrid.getBox(),STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.template applyStencils<Stencil10T>(sparseGrid.getBox(),STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();

        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = 2 * numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * Stencil01T::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = 2 * numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * Stencil01T::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << SparseGridZ::blockEdgeSize_ << "x" << SparseGridZ::blockEdgeSize_ << std::endl;
    std::cout << "Grid: " << gridEdgeSize*SparseGridZ::blockEdgeSize_ << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_ << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatGet_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    // typedef aggregate<float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN");

    testStencilHeatGet_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatGetZ_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    // typedef aggregate<float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilZ");

    testStencilHeatGet_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu_z<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilSkeleton_perf(unsigned int i, std::string base)
{
    auto testName = "In-place stencil";
    typedef SkeletonStencil<SparseGridZ::dims,0,1> Stencil01T;
    typedef SkeletonStencil<SparseGridZ::dims,1,0> Stencil10T;

    // typedef SkeletonStencil<SparseGridZ::dims,0,0> Stencil01T;
    // typedef SkeletonStencil<SparseGridZ::dims,0,0> Stencil10T;

    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*SparseGridZ::blockEdgeSize_);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize);
    dim3 blockSize(SparseGridZ::blockEdgeSize_,SparseGridZ::blockEdgeSize_);
    typename SparseGridZ::grid_info blockGeometry(gridSize);
    SparseGridZ sparseGrid(blockGeometry);
    gpu::ofp_context_t gpuContext;
    sparseGrid.template setBackgroundValue<0>(0);

    unsigned long long numElements = gridEdgeSize*SparseGridZ::blockEdgeSize_*gridEdgeSize*SparseGridZ::blockEdgeSize_;

    // Initialize the grid
    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
    sparseGrid.template flush < sRight_ < 0 >> (gpuContext, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    dim3 sourcePt(gridSize.x * SparseGridZ::blockEdgeSize_ / 2, gridSize.y * SparseGridZ::blockEdgeSize_ / 2, 0);
    insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
    sparseGrid.template flush < sRight_ < 0 >> (gpuContext, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

    iterations /= 2;
    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencils<Stencil01T>(sparseGrid.getBox(),STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.template applyStencils<Stencil10T>(sparseGrid.getBox(),STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();

        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = 2 * numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * Stencil01T::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = 2 * numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * Stencil01T::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << SparseGridZ::blockEdgeSize_ << "x" << SparseGridZ::blockEdgeSize_ << std::endl;
    std::cout << "Grid: " << gridEdgeSize*SparseGridZ::blockEdgeSize_ << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_ << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}

void launch_testConv3x3x3_perf_z_morton(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 3;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<8,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","Conv3x3x3");

    testConv3x3x3_perf<SparseGridGpu_z<dim, AggregateT, 8, chunkSize,long int>>("Convolution 3x3x3 Z-morton");
}

void launch_testConv3x3x3_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 3;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<8,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","Conv3x3x3");

    testConv3x3x3_perf<SparseGridGpu<dim, AggregateT, 8, chunkSize,long int>>("Convolution 3x3x3 ");
}

void launch_testConv3x3x3_perf_no_shared_z_morton(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 3;
    typedef aggregate<float,float> AggregateT;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","Conv3x3x3");

    testConv3x3x3_no_shared_perf<SparseGridGpu_z<dim, AggregateT, 8, 512, long int>>("Convolution 3x3x3_noshared z-morton");
}

void launch_testConv3x3x3_perf_no_shared(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 3;
    typedef aggregate<float,float> AggregateT;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","Conv3x3x3");

    testConv3x3x3_no_shared_perf<SparseGridGpu<dim, AggregateT, 8, 512, long int>>("Convolution 3x3x3_noshared");
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilSkeleton_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    // typedef aggregate<float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN");

    testStencilSkeleton_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}


template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilSkeletonZ_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    // typedef aggregate<float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilZ");

    testStencilSkeleton_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu_z<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}

BOOST_AUTO_TEST_CASE(testConv3x3x3_noshared)
{
    std::string testURI = suiteURI + ".device.conv3x3x3_no_shared.sparse.N.3D.gridScaling";
    unsigned int counter = 0;
    launch_testConv3x3x3_perf_no_shared(testURI, counter++);
    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testConv3x3x3_noshared_z_morton)
{
    std::string testURI = suiteURI + ".device.conv3x3x3_no_shared.sparse.N.3D.gridScaling";
    unsigned int counter = 0;
    launch_testConv3x3x3_perf_no_shared_z_morton(testURI, counter++);
    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testConv3x3x3)
{
    std::string testURI = suiteURI + ".device.conv3x3x3.sparse.N.3D.gridScaling";
    unsigned int counter = 0;
    launch_testConv3x3x3_perf(testURI, counter++);
    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testConv3x3x3_zmorton)
{

    std::string testURI = suiteURI + ".device.conv3x3x3_zmorton.sparse.N.3D.gridScaling";
    unsigned int counter = 0;
    launch_testConv3x3x3_perf_z_morton(testURI, counter++);
    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilSkeleton_gridScaling)
{
    std::string testURI = suiteURI + ".device.skeleton.dense.N.2D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilSkeleton_perf<blockEdgeSize, 128>(testURI, counter++);
    launch_testStencilSkeleton_perf<blockEdgeSize, 256>(testURI, counter++);
    launch_testStencilSkeleton_perf<blockEdgeSize, 512>(testURI, counter++);
    launch_testStencilSkeleton_perf<blockEdgeSize, 1024>(testURI, counter++);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatGet_gridScaling_2)
{
   std::string testURI = suiteURI + ".device.stencilGet.dense.N.2D.2.gridScaling";
   unsigned int counter = 0;
   launch_testStencilHeatGet_perf<2, 512>(testURI, counter++);
   launch_testStencilHeatGet_perf<2, 1024>(testURI, counter++);
   launch_testStencilHeatGet_perf<2, 2048>(testURI, counter++);
   // launch_testStencilHeatGet_perf<2, 4096>(testURI, counter++); // test

   testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatGet_gridScaling_4)
{
   std::string testURI = suiteURI + ".device.stencilGet.dense.N.2D.4.gridScaling";
   unsigned int counter = 0;
   launch_testStencilHeatGet_perf<4, 256>(testURI, counter++);
   launch_testStencilHeatGet_perf<4, 512>(testURI, counter++);
   launch_testStencilHeatGet_perf<4, 1024>(testURI, counter++);
   launch_testStencilHeatGet_perf<4, 2048>(testURI, counter++);

   testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatGet_gridScaling_8)
{
   std::string testURI = suiteURI + ".device.stencilGet.dense.N.2D.8.gridScaling";
   unsigned int counter = 0;
   launch_testStencilHeatGet_perf<8, 128>(testURI, counter++);
   launch_testStencilHeatGet_perf<8, 256>(testURI, counter++);
   launch_testStencilHeatGet_perf<8, 512>(testURI, counter++);
   launch_testStencilHeatGet_perf<8, 1024>(testURI, counter++);

   testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatGet_gridScaling_16)
{
   std::string testURI = suiteURI + ".device.stencilGet.dense.N.2D.16.gridScaling";
   unsigned int counter = 0;
   launch_testStencilHeatGet_perf<16, 64>(testURI, counter++);
   launch_testStencilHeatGet_perf<16, 128>(testURI, counter++);
   launch_testStencilHeatGet_perf<16, 256>(testURI, counter++);
   launch_testStencilHeatGet_perf<16, 512>(testURI, counter++);

   testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatGet_gridScaling_32)
{
   std::string testURI = suiteURI + ".device.stencilGet.dense.N.2D.32.gridScaling";
   unsigned int counter = 0;
   launch_testStencilHeatGet_perf<32, 32>(testURI, counter++);
   launch_testStencilHeatGet_perf<32, 64>(testURI, counter++);
   launch_testStencilHeatGet_perf<32, 128>(testURI, counter++);
   launch_testStencilHeatGet_perf<32, 256>(testURI, counter++); // test

   testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatGet_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencilGet.dense.N.2D.blockScaling";
    unsigned int counter = 0;
    // Note - blockEdgeSize == 2 doesn't work
    launch_testStencilHeatGet_perf<4, 2048>(testURI, counter++);
    launch_testStencilHeatGet_perf<8, 1024>(testURI, counter++);
    launch_testStencilHeatGet_perf<16, 512>(testURI, counter++);
    launch_testStencilHeatGet_perf<32, 256>(testURI, counter++);

    testSet.insert(testURI);
}


BOOST_AUTO_TEST_CASE(write_teport)
{
    write_test_report(report_sparsegrid_funcs, testSet);
}

BOOST_AUTO_TEST_SUITE_END()

BOOST_AUTO_TEST_SUITE_END()
